#define _GNU_SOURCE

#include <stdexcept>
#include <getopt.h>
#include <hip/hip_runtime.h>
#include <utility>
#include <vector>
#include <string_view>
#include <iostream>
#include <functional>
#include <error.h>
#include <sstream>
#include <unistd.h>

static const char usage[] = "[OPTION]... [FILTER]... <command>...";
static const char help[] = R"EOF(Options:
Options:
  -v, --verbose           Print progress of the devices through the filters.
  -h, --help              Print this help message.

Filters:
  -c, --capability [QUAL]MAJOR[.MINOR]
                          Only allow CUDA devices with the given compute capability.
                          QUAL can be any of =, <, >, all of which are inclusive.
  -r, --random=[N]        Allow at most N CUDA devices chosen at random. Defaults to 1.
)EOF";

using devlist = std::vector<std::pair<int, hipDeviceProp_t>>;

static devlist fetch_devices();
static void print_devlist(const devlist&);
static void filter_random(devlist&, const std::string&);
static void filter_capability(devlist&, const std::string&);

int main(int argc, char* const* argv) {
  if(getenv("CUDA_VISIBLE_DEVICES") != NULL) {
    std::cerr << "CUDA_VISIBLE_DEVICES is set, refusing to override\n";
    return 1;
  }

  devlist devs = fetch_devices();
  if(devs.empty()) {
    std::cerr << "No devices available!\n";
    return 77;  // SKIP
  }

  static const struct option opts[] = {
    {"help", no_argument, NULL, 'h'},
    {"capability", required_argument, NULL, 'c'},
    {"random", optional_argument, NULL, 'r'},
    {0, 0, 0, 0}
  };
  int opt;
  bool verbose = false;
  while((opt = getopt_long(argc, argv, "+hvc:r::", opts, nullptr)) != -1) {
    bool is_filter = false;
    switch(opt) {
    case 'h':
      std::cout << (argc > 0 ? argv[0] : "cuda-filter") << " " << usage << "\n\n" << help;
      return 0;
    case 'v':
      verbose = true;
      std::cerr << "Initial list of devices:\n";
      print_devlist(devs);
      break;
    case 'c':
      is_filter = true;
      if(verbose)
        std::cerr << "Applying --capability " << optarg << "\n";
      filter_capability(devs, optarg);
      break;
    case 'r':
      is_filter = true;
      if(verbose)
        std::cerr << "Applying --random " << (optarg ? optarg : "") << "\n";
      filter_random(devs, optarg ? optarg : "");
      break;
    default:
      std::cerr << "Usage: " << (argc > 0 ? argv[0] : "cuda-filter") << " " << usage << "\n";
      return 2;
    }

    if(is_filter) {
      if(verbose)
        print_devlist(devs);

      if(devs.empty()) {
        std::cerr << "No devices passed the filters!\n";
        return 77;  // SKIP
      }
    }
  }

  {
    std::ostringstream ss;
    bool first = true;
    for(const auto& dev: devs) {
      if(!first) ss << ",";
      first = false;
      ss << dev.first;
    }
    if(setenv("CUDA_VISIBLE_DEVICES", ss.str().c_str(), 1) != 0)
      error(1, errno, "Failed to setenv(CUDA_VISIBLE_DEVICES)");
  }

  std::vector<char*> new_argv;
  for(int i = optind; i < argc; i++) {
    new_argv.push_back(argv[i]);
  }
  if(new_argv.empty()) {
    std::cerr << "No command provided!\n";
    std::cerr << "Usage: " << (argc > 0 ? argv[0] : "cuda-filter") << " " << usage << "\n";
    return 2;
  }
  new_argv.push_back(NULL);

  execvp(new_argv[0], new_argv.data());
  error(127, errno, "Failed to exec");
  return 0;
}

devlist fetch_devices() {
  int nDevices = 0;
  hipError_t err = hipGetDeviceCount(&nDevices);
  if(err != hipSuccess || nDevices == 0)
    return {};

  devlist result;
  result.reserve(nDevices);
  for(int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, i);
    if(err != hipSuccess) {
      std::cerr << "Error fetching properties for device " << i << "\n";
      continue;
    }

    result.push_back({i, prop});
  }
  return result;
}

void print_devlist(const devlist& devs) {
  for(const auto& dev: devs)
    std::cerr << "  - " << dev.second.name << "\n";
}

void filter_random(devlist& devs, const std::string& arg) {
  int n = 1;
  if(!arg.empty())
    n = std::stoi(arg, nullptr, 10);
  if(n < 1) {
    std::cerr << "Invalid argument to --random, must be >= 1!\n";
    std::exit(2);
  }

  devlist bucket = std::move(devs);
  devs.clear();
  for(int i = 0; !bucket.empty() && i < n; i++) {
    int idx = nearbyint(drand48() * (bucket.size() - 1));
    devs.emplace_back(bucket.at(idx));
  }
}

void filter_capability(devlist& devs, const std::string& fullarg) {
  std::string_view arg = fullarg;

  std::function<bool(int, int)> qual = std::equal_to<int>();
  switch(arg[0]) {
  case '=':
    qual = std::equal_to<int>();
    arg = arg.substr(1);
    break;
  case '<':
    qual = std::less_equal<int>();
    arg = arg.substr(1);
    break;
  case '>':
    qual = std::greater_equal<int>();
    arg = arg.substr(1);
    break;
  }

  std::size_t pos;
  int major = std::stoi(std::string(arg), &pos, 10);
  arg = arg.substr(pos);

  int minor = -1;
  if(!arg.empty()) {
    if(arg[0] != '.') {
      std::cerr << "Invalid argument to --capability: " << fullarg;
      std::exit(2);
    }
    arg = arg.substr(1);
    minor = std::stoi(std::string(arg), nullptr, 10);
  }

  devlist bucket = std::move(devs);
  devs.clear();
  for(auto& dev: bucket) {
    if(qual(dev.second.major, major)
       && (minor == -1 || dev.second.major != major || qual(dev.second.minor, minor))) {
      devs.emplace_back(std::move(dev));
    }
  }
}
